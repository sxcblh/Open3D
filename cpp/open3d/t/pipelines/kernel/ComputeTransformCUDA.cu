#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hip/hip_runtime.h>

#include "open3d/core/CUDAUtils.h"
#include "open3d/core/Tensor.h"
#include "open3d/core/kernel/CUDALauncher.cuh"
#include "open3d/t/pipelines/kernel/ComputeTransformImpl.h"
#include "open3d/t/pipelines/kernel/Reduction6x6Impl.cuh"
#include "open3d/t/pipelines/kernel/TransformationConverter.h"

namespace open3d {
namespace t {
namespace pipelines {
namespace kernel {

const int kThread1DUnit = 256;

__global__ void ComputePosePointToPlaneCUDAKernel(
        const float *source_points_ptr,
        const float *target_points_ptr,
        const float *target_normals_ptr,
        const int64_t *correspondences_first,
        const int64_t *correspondences_second,
        const int n,
        float *global_sum) {
    __shared__ float local_sum0[kThread1DUnit];
    __shared__ float local_sum1[kThread1DUnit];
    __shared__ float local_sum2[kThread1DUnit];

    const int tid = threadIdx.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    const int workload_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (workload_idx >= n) return;

    float J[6] = {0}, reduction[21 + 6 + 2];
    float r = 0;

    bool valid = GetJacobianPointToPlane(workload_idx, source_points_ptr,
                                         target_points_ptr, target_normals_ptr,
                                         correspondences_first,
                                         correspondences_second, J, r);

    // Dump J, r into JtJ and Jtr
    int offset = 0;
    for (int i = 0; i < 6; ++i) {
        for (int j = 0; j <= i; ++j) {
            reduction[offset++] = J[i] * J[j];
        }
    }
    for (int i = 0; i < 6; ++i) {
        reduction[offset++] = J[i] * r;
    }
    reduction[offset++] = r * r;
    reduction[offset++] = valid;

    ReduceSum6x6LinearSystem<float, kThread1DUnit>(tid, valid, reduction,
                                                   local_sum0, local_sum1,
                                                   local_sum2, global_sum);
}

void ComputePosePointToPlaneCUDA(const float *source_points_ptr,
                                 const float *target_points_ptr,
                                 const float *target_normals_ptr,
                                 const int64_t *correspondences_first,
                                 const int64_t *correspondences_second,
                                 const int n,
                                 core::Tensor &pose,
                                 const core::Dtype &dtype,
                                 const core::Device &device) {
    core::Tensor global_sum =
            core::Tensor::Zeros({29}, core::Dtype::Float32, device);
    float *global_sum_ptr = global_sum.GetDataPtr<float>();

    const dim3 blocks((n + kThread1DUnit - 1) / kThread1DUnit);
    const dim3 threads(kThread1DUnit);

    ComputePosePointToPlaneCUDAKernel<<<blocks, threads, 0,
                                        core::cuda::GetStream()>>>(
            source_points_ptr, target_points_ptr, target_normals_ptr,
            correspondences_first, correspondences_second, n, global_sum_ptr);

    OPEN3D_CUDA_CHECK(hipDeviceSynchronize());

    // TODO (@rishabh), residual will be used for adding robust kernel support.
    float residual;
    int inlier_count;
    DecodeAndSolve6x6(global_sum, pose, residual, inlier_count);
}

}  // namespace kernel
}  // namespace pipelines
}  // namespace t
}  // namespace open3d
